#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer
// calculates the eye coordinate from user provided projection matrix 

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
typedef unsigned char VolumeType1;
typedef unsigned short VolumeType2;

texture<VolumeType/*BytesPerVoxel*/, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

__constant__ float c_sizeOfTransfertFunction;

typedef struct
{
    float4 m[3];
} float3x4;

typedef struct
{
    float4 m[4];
} float4x4;


__constant__ float4x4 c_invViewMatrix;  // inverse view matrix


__constant__ float4x4 c_invProjectionMatrix;  //  inverse projection matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;


    return smallest_tmax > largest_tmin;
}



// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}


__device__
float4 mul(const float4x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
	r.w = dot(v, M.m[3]);

    return r;
}

__device__
void printf4(const float4 &v)
{
  printf("kernel: %.2f  %.2f  %.2f  %.2f\n",v.x,v.y,v.z,v.w); 
}


__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

inline __device__ bool algoMaxProjection(float4 &acc, float4 &col )
{
       	acc = fmaxf(acc,col);
        
        return false ;
}

inline __device__ bool algoSumProjection(float4 &acc, float4 &col )
{
       	acc = fmaxf(acc,col);
        
        return false ;
}


inline __device__ bool algoBlendFrontToBack( float4 &acc, float4 &col )
{
        col *= col.w;
        // "over" operator for front-to-back blending
        acc = acc + col*(1.0f - acc.w);
        
        return false;
}

inline __device__ bool algoBlendBackToFront(float4 &acc, float4 &col )
{

        // "under" operator for back-to-front blending
        acc = lerp(acc, col, col.w);
        return false;
}

inline __device__ bool algo(float4 &acc, float4 &col )
{
		return algoMaxProjection(acc,col);
}




//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

extern "C" __global__ void
volumerender(uint *d_output, uint imageW, uint imageH,
		 			float scalex, float scaley, float scalez,
         	float brightness, float trangemin, float trangemax, float gamma)
{
		
    const int maxSteps = 512;
    const float tstep = 0.02f;
     
    const float ta = 1.0/(trangemax-trangemin);
    const float tb = trangemin/(trangemin-trangemax); 
    
    const float invscalex = 1/scalex;
    const float invscaley = 1/scaley;
    const float invscalez = 1/scalez;
    const float3 boxMin = make_float3(-scalex, -scaley, -scalez);
    const float3 boxMax = make_float3(scalex, scaley, scalez);

    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

	  	
    if ((x >= imageW) || (y >= imageH)) return;

    const float u = (x / (float) imageW)*2.0f-1.0f;
    const float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    float4 orig0, orig;
    float4 direc0, direc;
    float4 temp;
    float4 back,front;


   	front = make_float4(u,v,-1,1);
	back = make_float4(u,v,1,1);
  
    orig0 = mul(c_invProjectionMatrix,front);
	orig0 *= 1.f/orig0.w;
  
    orig = mul(c_invViewMatrix,orig0);
	orig *= 1.f/orig.w;
  

    direc0 = mul(c_invProjectionMatrix,back);
     
	direc0 *= 1.f/direc0.w;

	direc0 = normalize(direc0-orig0);


	direc = mul(c_invViewMatrix,direc0);
	direc.w = 0;

	 

	
    // calculate eye ray in world space
    Ray eyeRay;

	eyeRay.o = make_float3(orig);
	eyeRay.d = make_float3(direc);	
	


    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);


    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 acc = make_float4(0.0f);

	// float t = tnear;
    // float3 pos = eyeRay.o + eyeRay.d*tnear;
    // float3 step = eyeRay.d*tstep;


	float t = tnear;

	float4 pos;
	uint i;
	for(i=0; i<maxSteps; i++) {		
	  pos = orig + t*direc;


	  pos = pos*0.5f+0.5f;    // map position to [0, 1] coordinates

	  float sample = tex3D(tex, pos.x,pos.y,pos.z);
 
	  // Mapping to transfert function range and gamma correction: 
	  float mappedsample = powf(ta*sample+tb,gamma);
 
	  // lookup in transfer function texture
	  float4 col = tex1D(transferTex,mappedsample);
        
	  algo/*ProjectionAlgorythm*/(acc,col);

	  t += tstep;
	  if (t > tfar) break;
	}


    // for (int i=0; i<maxSteps; i++)
    // {
	  
    //     // read from 3D texture
    //     // remap position to [0, 1] coordinates
    //     float sample = tex3D(tex, invscalex*pos.x*0.5f+0.5f, invscaley*pos.y*0.5f+0.5f, invscalez*pos.z*0.5f+0.5f);
 
 	// 			// Mapping to transfert function range and gamma correction: 
 	// 			float mappedsample = powf(ta*sample+tb,gamma);
 
    //     // lookup in transfer function texture
    //     float4 col = tex1D(transferTex,mappedsample);
        
    //     algo/*ProjectionAlgorythm*/(acc,col);

    //     t += tstep;

    //     if (t > tfar) break;
    //     pos += step;
    // }
    
    acc *= brightness;
    
    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(acc);
}


/*
extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float scalex, float scaley, float scalez, float brightness, float trangemin, float trangemax, float gamma)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, scalex, scaley, scalez, 
                                      brightness, trangemin, trangemax, gamma);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}
/**/


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
