#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
  Adapted from the Nvidia CUDA samples
  http://developer.download.nvidia.com/compute/cuda/4_2/rel/sdk/website/OpenCL/html/samples.html
 

  Authors: 	Loic Royer		 (royer@mpi-cbg.de)
  					Martin Weigert (mweigert@mpi-cbg.de)
 */

// Simple 3D volume renderer
// calculates the eye coordinate from user provided projection matrix 


#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


// Loop unrolling length:
#define LOOPUNROLL 16

// Typedefs:
typedef unsigned int  uint;
typedef unsigned char uchar;
typedef unsigned char VolumeType;
typedef unsigned char VolumeType1;
typedef unsigned short VolumeType2;

typedef struct
{
    float4 m[3];
} float3x4;

typedef struct
{
    float4 m[4];
} float4x4;

// Arrays:
hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

// Textures:
texture<VolumeType/*BytesPerVoxel*/, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

// Constants holding matrices:
__constant__ float c_sizeOfTransfertFunction;
__constant__ float4x4 c_invViewMatrix;  // inverse view matrix
__constant__ float4x4 c_invProjectionMatrix;  //  inverse projection matrix

// Ray structure:
struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};


// random number generator for dithering
__forceinline__
__device__
float random(uint x, uint y)
{   
    uint a = 4421 +(1+x)*(1+y) +x +y;
    for(int i=0; i < 10; i++)
    {
        a = (uint(1664525) * a + uint(1013904223)) % uint(79197919);
    }
    float rnd = (a*1.0)/(79197919.f);
    return rnd-0.5f;
}


// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm
__forceinline__
__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;


    return smallest_tmax > largest_tmin;
}


// transform vector by matrix (no translation)
__forceinline__
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__forceinline__
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

// transform vector by matrix with translation:
__forceinline__
__device__
float4 mul(const float4x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
		r.w = dot(v, M.m[3]);

    return r;
}


// convert float4 into uint:
__forceinline__ 
__device__ 
uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}


// convert float4 into uint and take the max with an existing RGBA value in uint form:
__forceinline__ 
__device__ 
uint rgbaFloatToIntAndMax(uint existing, float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    
    const uint nr = uint(rgba.x*255);
    const uint ng = uint(rgba.y*255);
    const uint nb = uint(rgba.z*255);
    const uint na = uint(rgba.w*255);
    
    const uint er = existing&0xFF;
    const uint eg = (existing>>8)&0xFF;
    const uint eb = (existing>>16)&0xFF;
    const uint ea = (existing>>24)&0xFF;
    
    const uint  r = max(nr,er);
    const uint  g = max(ng,eg);
    const uint  b = max(nb,eb);
    const uint  a = max(na,ea);
    
    return a<<24|b<<16|g<<8|r ;
}



// Render function,
// performs max projection and then uses the transfert function to obtain a color per pixel:
extern "C" __global__ void
volumerender(       uint *d_output, 
							const uint imageW, 
							const uint imageH,
							const float brightness, 
							const float trangemin, 
							const float trangemax, 
							const float gamma, 
							const int   maxsteps,
							const float dithering,
							const float phase,
							const int   clear)
{
		
		// convert range bounds to linear map:
    const float ta = 1.0f/(trangemax-trangemin);
    const float tb = trangemin/(trangemin-trangemax); 
    
   	// box bounds:
    const float3 boxMin = make_float3(-1.f, -1.f, -1.f);
    const float3 boxMax = make_float3(1.f,1.f,1.f);

		// thread int coordinates:
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

		// thread float coordinates:
    const float u = (x / (float) imageW)*2.0f-1.0f;
    const float v = (y / (float) imageH)*2.0f-1.0f;

		// Back and front before all transformations.
   	const float4 front = make_float4(u,v,-1.f,1.f);
		const float4 back = make_float4(u,v,1.f,1.f);

    // calculate eye ray in world space
    float4 orig0, orig;
    float4 direc0, direc;
  
  	// Origin point
    orig0 = mul(c_invProjectionMatrix,front);
		orig0 *= 1.f/orig0.w;
    orig = mul(c_invViewMatrix,orig0);
		orig *= 1.f/orig.w;
  
  	// Direction:
    direc0 = mul(c_invProjectionMatrix,back);
		direc0 *= 1.f/direc0.w;
		direc0 = normalize(direc0-orig0);
		direc = mul(c_invViewMatrix,direc0);
		direc.w = 0.0f;

    // calculate eye ray in world space
    Ray eyeRay;
		eyeRay.o = make_float3(orig);
		eyeRay.d = make_float3(direc);	
	
    // find intersection with box
    float tnear, tfar;
    const int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) 
    {
	  	d_output[x+imageW*y] = 0.f;
	    return;
    }

    // clamp to near plane:
		if (tnear < 0.0f) tnear = 0.0f;     

		// compute stape size:		
		const float tstep = abs(tnear-tfar)/maxsteps;

	  //if ((u == 0f) && (v == 0f))
    //  printf("tnear=%f tfar=%f diff=%f tstep=%f ratio=%f \n",tnear,tfar, abs(tnear-tfar),tstep, abs(tnear-tfar)/tstep);

		// apply phase:
		orig += phase*tstep*direc;

		// randomize origin point a bit:
		const uint entropy = (uint)( 6779514*length(orig) + 6257327*length(direc) );
		orig += dithering*tstep*random(entropy+x,entropy+y)*direc;
		
		// precompute vectors: 
		const float4 vecstep = 0.5f*tstep*direc;
		float4 pos = orig*0.5f+0.5f + tnear*0.5f*direc;

		// Loop unrolling setup: 
    const uint unrolledmaxsteps = (maxsteps/LOOPUNROLL)+1;
		
		// raycasting loop:
		float maxp = 0.0f;
		for(int i=0; i<unrolledmaxsteps; i++) 
		{
			for(int j=1; j<LOOPUNROLL; j++)
			{
		  	maxp = fmaxf(maxp,tex3D(tex, pos.x,pos.y,pos.z));
		  	pos+=vecstep;
		  }
		}
		
		// Mapping to transfert function range and gamma correction: 
		const float mappedsample = __saturatef(powf(ta*maxp+tb,gamma));
	 
		// lookup in transfer function texture:
		const float4 color = brightness * tex1D(transferTex,mappedsample);

    
    // write output color:
    d_output[y*imageW + x] = rgbaFloatToIntAndMax(clear*d_output[y*imageW + x],color);
}

