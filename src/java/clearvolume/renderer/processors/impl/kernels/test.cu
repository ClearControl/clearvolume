#include "hip/hip_runtime.h"

// Dumb unoptimized summation kernel - for demonstration purposes


typedef unsigned char VolumeType;
typedef unsigned char VolumeType1;
typedef unsigned short VolumeType2;

texture<VolumeType/*BytesPerVoxel*/, 3, hipReadModeNormalizedFloat> tex;


extern "C" __global__ void test()
{
	if(blockIdx.x == 0 && threadIdx.x==0)
		printf("value=%f\n", tex3D(tex, 10, 10, 10));
}


